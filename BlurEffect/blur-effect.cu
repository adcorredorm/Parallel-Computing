#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#define PGMHeaderSize           0x40

inline bool loadPPM(const char *file, unsigned char **data, unsigned int *w, unsigned int *h, unsigned int *channels)
{
    FILE *fp = NULL;

    fp = fopen(file, "rb");
         if (!fp) {
              fprintf(stderr, "__LoadPPM() : unable to open file\n" );
                return false;
         }

    // check header
    char header[PGMHeaderSize];

    if (fgets(header, PGMHeaderSize, fp) == NULL)
    {
        fprintf(stderr,"__LoadPPM() : reading PGM header returned NULL\n" );
        return false;
    }

    if (strncmp(header, "P5", 2) == 0)
    {
        *channels = 1;
    }
    else if (strncmp(header, "P6", 2) == 0)
    {
        *channels = 3;
    }
    else
    {
        fprintf(stderr,"__LoadPPM() : File is not a PPM or PGM image\n" );
        *channels = 0;
        return false;
    }

    // parse header, read maxval, width and height
    unsigned int width = 0;
    unsigned int height = 0;
    unsigned int maxval = 0;
    unsigned int i = 0;

    while (i < 3)
    {
        if (fgets(header, PGMHeaderSize, fp) == NULL)
        {
            fprintf(stderr,"__LoadPPM() : reading PGM header returned NULL\n" );
            return false;
        }

        if (header[0] == '#')
        {
            continue;
        }

        if (i == 0)
        {
            i += sscanf(header, "%u %u %u", &width, &height, &maxval);
        }
        else if (i == 1)
        {
            i += sscanf(header, "%u %u", &height, &maxval);
        }
        else if (i == 2)
        {
            i += sscanf(header, "%u", &maxval);
        }
    }

    // check if given handle for the data is initialized
    if (NULL != *data)
    {
        if (*w != width || *h != height)
        {
            fprintf(stderr, "__LoadPPM() : Invalid image dimensions.\n" );
        }
    }
    else
    {
        *data = (unsigned char *) malloc(sizeof(unsigned char) * width * height * *channels);
        if (!data) {
         fprintf(stderr, "Unable to allocate hostmemory\n");
         return false;
        }
        *w = width;
        *h = height;
    }

    // read and close file
    if (fread(*data, sizeof(unsigned char), width * height * *channels, fp) == 0)
    {
        fprintf(stderr, "__LoadPPM() : read data returned error.\n" );
        fclose(fp);
        return false;
    }

    fclose(fp);

    return true;
}

inline bool savePPM(const char *file, unsigned char *data, unsigned int w, unsigned int h, unsigned int channels)
{
    assert(NULL != data);
    assert(w > 0);
    assert(h > 0);

    std::fstream fh(file, std::fstream::out | std::fstream::binary);

    if (fh.bad())
    {
        fprintf(stderr, "__savePPM() : Opening file failed.\n" );
        return false;
    }

    if (channels == 1)
    {
        fh << "P5\n";
    }
    else if (channels == 3)
    {
        fh << "P6\n";
    }
    else
    {
        fprintf(stderr, "__savePPM() : Invalid number of channels.\n" );
        return false;
    }

    fh << w << "\n" << h << "\n" << 0xff << std::endl;

    for (unsigned int i = 0; (i < (w*h*channels)) && fh.good(); ++i)
    {
        fh << data[i];
    }

    fh.flush();

    if (fh.bad())
    {
        fprintf(stderr,"__savePPM() : Writing data failed.\n" );
        return false;
    }

    fh.close();

    return true;
}

#define TILE      16
int kernel_size;
unsigned char *data=NULL, *d_input=NULL, *d_output=NULL;
unsigned int w ,h ,channels;


__global__ void box_filter(const unsigned char *in, unsigned char *out, const unsigned int w, const unsigned int h, int kernel_size, const int block){
    //Indexes
    const int R = (kernel_size - 1)/2;
    const int x = blockIdx.x * TILE + threadIdx.x - R;       // x image index
    const int y = blockIdx.y * TILE + threadIdx.y - R;       // y image index
    const int d = (y * w + x) * 3 ;                          // red pixel index
    //shared mem
    
    __shared__ float shMem[30][30][3];
    if(x<0 || y<0 || x>=w || y>=h) {            
        shMem[threadIdx.x][threadIdx.y][0] = 0;
        shMem[threadIdx.x][threadIdx.y][1] = 0;
        shMem[threadIdx.x][threadIdx.y][2] = 0;
        return; 
    }
    shMem[threadIdx.x][threadIdx.y][0] = in[d];
    shMem[threadIdx.x][threadIdx.y][1] = in[d+1];
    shMem[threadIdx.x][threadIdx.y][2] = in[d+2];
    
    __syncthreads();

    if ((threadIdx.x >= R) && (threadIdx.x < (block-R)) && (threadIdx.y >= R) && (threadIdx.y < (block-R))) {
        float red = 0;
        float green = 0;
        float blue = 0;
        for(int dx=-R; dx<=R; dx++) {
            for(int dy=-R; dy<=R; dy++) {
                red += shMem[threadIdx.x+dx][threadIdx.y+dy][0];
                green += shMem[threadIdx.x+dx][threadIdx.y+dy][1];
                blue += shMem[threadIdx.x+dx][threadIdx.y+dy][2];
            }
        }
    out[d] = red / (kernel_size * kernel_size);
    out[d+1] = green / (kernel_size * kernel_size);
    out[d+2] = blue / (kernel_size * kernel_size);
    
    }
}


#define checkCudaErrors(err) __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void procces_image(){
    const int R = (kernel_size - 1)/2;
    const int block = TILE+(2*R);

    size_t size = w*h*channels * sizeof(unsigned char);

    // Device Malloc
    checkCudaErrors(hipMalloc((void **)&d_input, size));
    checkCudaErrors(hipMalloc((void **)&d_output, size));

    // Copy to device
    checkCudaErrors(hipMemcpy(d_input, data, size, hipMemcpyHostToDevice));

    int GRID_W = w/TILE +1;
    int GRID_H = h/TILE +1;
    dim3 threadsPerBlock(block, block);
    dim3 blocksPerGrid(GRID_W,GRID_H);
    

    // Launch Kernel
    printf("CUDA kernel launch with [%d %d] blocks of [%d %d] threads\n", blocksPerGrid.x, blocksPerGrid.y, threadsPerBlock.x, threadsPerBlock.y);
    box_filter<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, w, h, kernel_size, block);
   
    checkCudaErrors(hipGetLastError());

    // Copy data from device to host
    checkCudaErrors(hipMemcpy(data, d_output, size, hipMemcpyDeviceToHost));

}

int main(int argc, char *argv[]){
    if(argc != 4) abort();
    kernel_size = atoi(argv[3]);
    struct timeval tval_before, tval_after, tval_result;

    if(! loadPPM(argv[1], &data, &w, &h, &channels)){
        fprintf(stderr, "Failed to open File\n");
        exit(EXIT_FAILURE);
    }

    gettimeofday(&tval_before, NULL);

    procces_image();

    gettimeofday(&tval_after, NULL);

    // Save Picture
    if (!savePPM(argv[2], data, w,  h,  channels)){
        fprintf(stderr, "Failed to save File\n");
        exit(EXIT_FAILURE);
    }

    // Free memory
    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
    free(data);

    timersub(&tval_after,&tval_before,&tval_result);
    printf("-\tTime elapsed: %ld.%06lds\t-\n",  (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
//    printf("%ld.%06ld;",  (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
}